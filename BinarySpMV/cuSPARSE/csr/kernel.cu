#include "hip/hip_runtime.h"
//Example: Application using C++ and the CUSPARSE library
//-------------------------------------------------------
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <ctype.h>
#include <string.h>
#include <math.h>
#include <cfloat>
#include <errno.h>
#include "hipsparse.h"
#include "mmio.h"
#include <omp.h>
#include <dirent.h>

using namespace std;

#define NUM_TRANSFER 50
#define NUM_RUN 100

inline void checkcuda(hipError_t result)
{
	if (result != hipSuccess) {
		printf("CUDA Runtime Error: %s\n", hipGetErrorString(result));
		printf("hello");
	}
}

inline void checkcusparse(hipsparseStatus_t result)
{
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		printf("CUSPARSE Error, error_code =  %d\n", result);
	}
}

double average(int n, double *data)
{
    double ave = 0.0;
    for(int i = 0; i < n; i++)
        ave += data[i];
    
    return ave / n;
}

double variance(int n, double ave, double *data)
{
    double var = 0.0;
    for (int i = 0; i < n; i++) {
        double temp = data[i] - ave;
        var += (temp * temp);
    }
    
    return var / n;
}

int readMtx(char *filename, int &m, int &n, int &nnzA, int *&csrRowIdxA, int *&csrColIdxA,
	float *&csrValA);
int cudaSpmv(int *&csrRowIndexHostPtr, int *&csrColIndexHostPtr,
			 float *&csrValHostPtr, float *&xHostPtr, float *&yHostPtr);
void queryDevice();

/*** Declaration ***/
int M, N, nnz;				 // M (row number), N (column number), nnz (Number of Non-Zero members)
char matrixName[1024] = {0}; 

// host variables
int *csrRowIndexHostPtr = 0; // coo format row index
int *csrColIndexHostPtr = 0; // coo format column index
float *csrValHostPtr = 0;	 // coo format value index
float *xHostPtr = 0;		 // the multiplied vector
float *yHostPtr = 0;		 // the result vector

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		printf("usage: ./exe MatrixFile\n");
		return 0;
	}

	printf("%s %s\n", argv[0], argv[1]);

	queryDevice();

	char matrix_dir[1024] = {0};
	strcpy(matrix_dir, argv[1]);

	//find matrix file
	DIR *matrix_dir_handle;
	struct dirent *matrix_entry;
	matrix_dir_handle = opendir(matrix_dir);

	int counter = 0, error_count = 0;
	while ((matrix_entry = readdir(matrix_dir_handle)) != NULL) {
		if (strcmp(matrix_entry->d_name, "..") != 0 && strcmp(matrix_entry->d_name, ".") != 0) {
			char source[1024] = {0};
			strcpy(source, argv[1]);
			strcat(source, "/");
			strcat(source, matrix_entry->d_name);

			strcpy(matrixName, matrix_entry->d_name);
			printf("%s\n", matrixName);

			//deal every matrix. source is the complete path name of the matrix
			readMtx(source, M, N, nnz, csrRowIndexHostPtr, csrColIndexHostPtr, csrValHostPtr);

			xHostPtr = (float *)malloc(N * sizeof(float));
			for (int i = 0; i < N; i++) xHostPtr[i] = 1.0;
			yHostPtr = (float *)malloc(M * sizeof(float));

			cudaSpmv(csrRowIndexHostPtr, csrColIndexHostPtr, csrValHostPtr, xHostPtr, yHostPtr);

			float * y_ref = (float *)malloc(sizeof(float) * M);
			for (int i = 0; i < M; i++)
			{
				float sum = 0.0;
				for (int j = csrRowIndexHostPtr[i]; j < csrRowIndexHostPtr[i+1]; j++)
					sum += xHostPtr[csrColIndexHostPtr[j]] * csrValHostPtr[j];
				y_ref[i] = sum;
			}

			for (int i = 0; i < M; i++)
				if ( abs(y_ref[i] - yHostPtr[i]) > 1e-6 ) {
					error_count++;
					break;
				}

			free(csrRowIndexHostPtr);
			free(csrColIndexHostPtr);
			free(csrValHostPtr);
			free(xHostPtr);
			free(yHostPtr);
			free(y_ref);
			counter++;
			printf("%d finished, %d error\n\n\n",counter, error_count);
		} //end if
	}	  //end while

	printf("\nTest finished!  %d matrices has been test, error_count = %d\n", counter, error_count);
	printf("--------------------------------------------------\n\n");

	return 0;
}

int readMtx(char *filename, int &m, int &n, int &nnzA, int *&csrRowPtrA, int *&csrColIdxA,
			 float *&csrValA)
{
	int ret_code;
	MM_typecode matcode;

	FILE *f = NULL;
	int nnzA_mtx_report;
	int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;
	// load matrix
	if ((f = fopen(filename, "r")) == NULL)
		return -1;

	if (mm_read_banner(f, &matcode) != 0) {
		printf("Could not process Matrix Market banner.\n");
		return -2;
	}

	if (mm_is_complex(matcode)) {
		printf("Sorry, data type 'COMPLEX' is not supported. \n");
		return -3;
	}

	if (mm_is_pattern(matcode)) {
		isPattern = 1; printf("type = Pattern.\n");
	}

	if (mm_is_real(matcode)) {
		isReal = 1; printf("type = real.\n");
	}

	if (mm_is_integer(matcode)) {
		isInteger = 1; printf("type = integer.\n");
	}

	ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
	if (ret_code != 0)
		return -4;

	if (mm_is_symmetric(matcode) || mm_is_hermitian(matcode)) {
		isSymmetric = 1;
		printf("symmetric = true.\n");
	}
	else {
		printf("symmetric = false.\n");
	}

	int *csrRowPtrA_counter = (int *)malloc((m + 1) * sizeof(int));
	memset(csrRowPtrA_counter, 0, (m + 1) * sizeof(int));

	int *csrRowIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
	int *csrColIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
	float *csrValA_tmp = (float *)malloc(nnzA_mtx_report * sizeof(float));

	for (int i = 0; i < nnzA_mtx_report; i++)
	{
		int idxi, idxj;
		double fval;
		int ival;

		if (isReal)
			fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
		else if (isInteger)
		{
			fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
			fval = ival;
		}
		else if (isPattern)
		{
			fscanf(f, "%d %d\n", &idxi, &idxj);
			fval = 1.0;
		}

		// adjust from 1-based to 0-based
		idxi--;
		idxj--;

		csrRowPtrA_counter[idxi]++;
		csrRowIdxA_tmp[i] = idxi;
		csrColIdxA_tmp[i] = idxj;
		csrValA_tmp[i] = fval;
	}

	if (f != stdin)
		fclose(f);

	if (isSymmetric)
	{
		for (int i = 0; i < nnzA_mtx_report; i++)
		{
			if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
				csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
		}
	}

	// exclusive scan for csrRowPtrA_counter
	int old_val, new_val;

	old_val = csrRowPtrA_counter[0];
	csrRowPtrA_counter[0] = 0;
	for (int i = 1; i <= m; i++)
	{
		new_val = csrRowPtrA_counter[i];
		csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i - 1];
		old_val = new_val;
	}

	nnzA = csrRowPtrA_counter[m];
	csrRowPtrA = (int *)malloc((m + 1) * sizeof(int));
	memcpy(csrRowPtrA, csrRowPtrA_counter, (m + 1) * sizeof(int));
	memset(csrRowPtrA_counter, 0, (m + 1) * sizeof(int));

	csrColIdxA = (int *)malloc(nnzA * sizeof(int));
	csrValA = (float *)malloc(nnzA * sizeof(float));

	if (isSymmetric)
	{
		for (int i = 0; i < nnzA_mtx_report; i++)
		{
			if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
			{
				int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
				csrColIdxA[offset] = csrColIdxA_tmp[i];
				csrValA[offset] = csrValA_tmp[i];
				csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

				offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
				csrColIdxA[offset] = csrRowIdxA_tmp[i];
				csrValA[offset] = csrValA_tmp[i];
				csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
			}
			else
			{
				int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
				csrColIdxA[offset] = csrColIdxA_tmp[i];
				csrValA[offset] = csrValA_tmp[i];
				csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
			}
		}
	}
	else
	{
		for (int i = 0; i < nnzA_mtx_report; i++)
		{
			int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
			csrColIdxA[offset] = csrColIdxA_tmp[i];
			csrValA[offset] = csrValA_tmp[i];
			csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
		}
	}

	// free tmp space
	free(csrColIdxA_tmp);
	free(csrValA_tmp);
	free(csrRowIdxA_tmp);
	free(csrRowPtrA_counter);
	
	return 0;
}

int cudaSpmv(int *&csrRowIndexHostPtr, int *&csrColIndexHostPtr, float *&csrValHostPtr, float *&xHostPtr, float *&yHostPtr)
{
	// GPU variables
	hipsparseHandle_t handle = 0;  // cusparse handle, if you want to use cusparse ,you must create a cusparse handle
	hipsparseMatDescr_t descr = 0; // a matrix descriptor used for multiplication
	float done = 1.0;			  //float number 1
	float dzero = 0.0;			  //float number 0
	int memory_size = 0;

	int *csrRowIndexGpuPtr = 0;	  // coo format row index
	int *csrColIndexGpuPtr = 0;	  // coo format column index
	float *csrValGpuPtr = 0;	  // coo format value index
	float *xGpuPtr = 0;			  // the multiplied vector
	float *yGpuPtr = 0;			  // the result vector

	/*** Allocate GPU Memory ***/
	checkcuda(hipMalloc((void **)&csrRowIndexGpuPtr, (M+1) * sizeof(int)));
	checkcuda(hipMalloc((void **)&csrColIndexGpuPtr, nnz * sizeof(int)));
	checkcuda(hipMalloc((void **)&csrValGpuPtr, nnz * sizeof(float)));
	checkcuda(hipMalloc((void **)&xGpuPtr, N * sizeof(float)));
	checkcuda(hipMalloc((void **)&yGpuPtr, M * sizeof(float)));

	memory_size += nnz * 2;		  // memory of csrValGpuPtr, csrColIndexGpuPtr
	memory_size += M + 1; // memory of csrRowIndexGpuPtr
	printf("memory_size = %.4fMB.\n", (float)(memory_size)*4 / 1024 / 1024);

	double transfer_time = 0.0, calculate_time = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start); // create event
	hipEventCreate(&stop);

	// initialize cusparse library
	checkcusparse(hipsparseCreate(&handle));
	// initialize matrix descriptor
	checkcusparse(hipsparseCreateMatDescr(&descr));
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	// copy x vector from CPU to GPU
	checkcuda(hipMemcpy(xGpuPtr, xHostPtr, (N * sizeof(float)), hipMemcpyHostToDevice));

	double all_transfer_time[NUM_TRANSFER];
	for (int i = 0; i < NUM_TRANSFER; i++) {
        float temp = 0.0;
        hipEventRecord(start, 0);
		// copy CSR-based matrix from CPU to GPU
		checkcuda(hipMemcpy(csrRowIndexGpuPtr, csrRowIndexHostPtr, ((M + 1) * sizeof(int)), hipMemcpyHostToDevice));
		checkcuda(hipMemcpy(csrColIndexGpuPtr, csrColIndexHostPtr, (nnz * sizeof(int)), hipMemcpyHostToDevice));
		checkcuda(hipMemcpy(csrValGpuPtr, csrValHostPtr, (nnz * sizeof(float)), hipMemcpyHostToDevice));
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&temp, start, stop);
        all_transfer_time[i] = temp;
	}
    transfer_time = average(NUM_TRANSFER, all_transfer_time);
    double var_transfer_time = variance(NUM_TRANSFER, transfer_time, all_transfer_time);
	printf("%s transmission time %.6f ", matrixName, transfer_time);

    /*** run one time for checking result ***/
    checkcuda(hipMemset(yGpuPtr, 0.0, M * sizeof(float)));
    checkcusparse(hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, nnz, &done, descr, csrValGpuPtr, csrRowIndexGpuPtr, csrColIndexGpuPtr, xGpuPtr, &dzero, yGpuPtr));
    checkcuda(hipMemcpy(yHostPtr, yGpuPtr, (M * sizeof(float)), hipMemcpyDeviceToHost));
	
    double all_spmv_time[NUM_RUN];
	for (int i = 0; i < NUM_RUN; i++)
	{
        float temp = 0.0;
        hipEventRecord(start, 0); 
		/*** CSR SpMV ***/
		checkcusparse(hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, nnz, &done, descr, csrValGpuPtr, csrRowIndexGpuPtr, csrColIndexGpuPtr, xGpuPtr, &dzero, yGpuPtr));
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&temp, start, stop);
        all_spmv_time[i] = temp;
    }
	calculate_time = average(NUM_RUN, all_spmv_time);
    double var_calculate_time = variance(NUM_RUN, calculate_time, all_spmv_time);
	printf("calculation time %.6f\n", calculate_time);

	// FILE *fresult = fopen("cusparse_csr_time_float_V100.txt", "a+");
	// if (fresult == NULL) {
	// 	printf("Create file failed.\n ");
	// }
	// else {
	// 	fprintf(fresult, "%s %.6f %.6f %.6f %.6f\n", matrixName, transfer_time, var_transfer_time, calculate_time, var_calculate_time);
	// 	fclose(fresult);
	// }

	/*** Release Resource ***/
	// destroy matrix descriptor
	checkcusparse(hipsparseDestroyMatDescr(descr));
	descr = 0;
	// destroy handle
	checkcusparse(hipsparseDestroy(handle));
	handle = 0;
	hipFree(csrRowIndexGpuPtr);
	hipFree(csrColIndexGpuPtr);
	hipFree(csrValGpuPtr);
	hipFree(xGpuPtr);
	hipFree(yGpuPtr);
	return 0;
}

void queryDevice()
{
	hipDeviceProp_t deviceProp;
	int deviceCount = 0;
	hipError_t hipError_t;
	hipError_t = hipGetDeviceCount(&deviceCount);
	cout << "hipError_t = " << hipError_t << endl;
	for (int i = 0; i < deviceCount; i++)
	{
		hipError_t = hipGetDeviceProperties(&deviceProp, i);
		cout << "Device " << i << endl;
		cout << "Device name: " << deviceProp.name << endl;
		cout << "Total global memory (MB) : " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;
		cout << "Share memory per block (KB) : " << deviceProp.sharedMemPerBlock / 1024 << endl;
		cout << "Number of registers per block (KB) : " << deviceProp.regsPerBlock << endl;
		cout << "Maximum threads per block : " << deviceProp.maxThreadsPerBlock << endl;
		cout << "Compute capability : " << deviceProp.major << "." << deviceProp.minor << endl;
		cout << "Number of multi-processor : " << deviceProp.multiProcessorCount << endl;
	}
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		printf("hipSetDevice failed!");

	int device = -1;
	cudaStatus = hipGetDevice(&device);
	if (cudaStatus != hipSuccess)
		printf("hipGetDevice failed!");
	cout << "\nThe device now beening used is device " << device << endl;
}
